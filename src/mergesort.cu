#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================

 COMPILAR USANDO O SEGUINTE COMANDO:

 nvcc segmented_sort.cu -o segmented_sort -std=c++11 --expt-extended-lambda -I"/home/schmid/Dropbox/Unicamp/workspace/sorting_segments/moderngpu-master/src"

 */
#include <moderngpu/kernel_mergesort.hxx>

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <utility>
#include <iostream>
#include <bitset>
#include <math.h>

#include <hip/hip_runtime.h>
#include <iostream>

#include<thread>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 16
#endif

//using namespace mgpu;
using namespace std;
using namespace std::placeholders;

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(uint* host_data, uint n, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << host_data[i * m + j] << " ";
		}
		std::cout << "\n";
	}

}

////////////////////////////////////////////////////////////////////////////////
// Verify the results.
////////////////////////////////////////////////////////////////////////////////
void check_results(int n, int m, unsigned int *results_h) {
	for (int i = 0; i < n; ++i) {
		for (uint j = 1; j < m; j++) {
			if (results_h[i * m + j - 1] > results_h[i * m + j]) {
				std::cout << "Invalid item[" << j - 1 << "]: "
						<< results_h[i * m + j - 1] << " greater than "
						<< results_h[i * m + j] << std::endl;
				exit (EXIT_FAILURE);
			}
		}
	}

	std::cout << "OK" << std::endl;
}

/*class Teste {
 uint* d_vec;
 uint* h_seg;
 uint num_of_elements;
 uint num_of_segments;
 uint nstreams;
 public:
 Teste(uint* d_vec, uint* h_seg, uint num_of_elements, uint num_of_segments, uint nstreams){
 this->d_vec = d_vec;
 this->h_seg = h_seg;
 this->num_of_elements = num_of_elements;
 this->num_of_segments = num_of_segments;
 this->nstreams = nstreams;
 }

 void create(){
 std::thread t1(&Teste::sorting);
 t1.join();
 }

 void exec() {
 //t1.();
 }

 void sorting() {
 //hipEventRecord(start);
 for(int i = 0; i < num_of_segments; i+=nstreams) {
 for (int s = 0; s < nstreams; s++) {

 }
 }
 //hipEventRecord(stop);
 }
 };
 void sorting(uint* d_vec, uint segmentid, uint elements, mgpu::stream_context_t context) {
 mgpu::mergesort(d_vec + segmentid, elements, mgpu::less_t<uint>(), context);
 }*/

int main(int argc, char** argv) {

	uint num_of_segments;
	uint num_of_elements;
	uint i;

	scanf("%d", &num_of_segments);
	uint mem_size_seg = sizeof(uint) * (num_of_segments);
	uint *h_seg = (uint *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	uint mem_size_vec = sizeof(uint) * num_of_elements;
	uint *h_vec = (uint *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++)
		scanf("%d", &h_vec[i]);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_vec;

	hipStream_t streams[NUM_STREAMS];
	mgpu::stream_context_t contexts[NUM_STREAMS];
	for (int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&streams[i]);
		contexts[i].setStream(streams[i]);
	}

	int nstreams = NUM_STREAMS;
	if (NUM_STREAMS > num_of_segments)
		nstreams = num_of_segments;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));

	for (uint j = 0; j < EXECUTIONS; j++) {

		// copy host memory to device
		cudaTest(
				hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));
		//Teste t(d_vec, h_seg, num_of_elements, num_of_segments, nstreams);
		//t.create();
		try {
			hipEventRecord(start);
			for (int i = 0; i < num_of_segments; i += nstreams) {
				for (int s = 0; s < nstreams; s++) {
					//mgpu::mergesort(d_vec+h_seg[i+s], num_of_elements/num_of_segments, mgpu::less_t<uint>(), contexts[s]);
					//std::thread t1(sorting, d_vec, h_seg[i+s], num_of_elements/num_of_segments, contexts[s]);
					//mgpu::mergesort(d_vec + h_seg[i + s],	num_of_elements / num_of_segments, mgpu::less_t<uint>(), contexts[s]);
					mgpu::mergesort<mgpu::empty_t, uint, mgpu::less_t<uint>>
					(d_vec+h_seg[i+s], h_seg[i+s+1]-h_seg[i+s], mgpu::less_t<uint>(), contexts[s]);
					//std::thread t1([=] { mgpu::mergesort<mgpu::empty_t, uint, mgpu::less_t<uint>>
						//(d_vec+h_seg[i+s], num_of_elements/num_of_segments, mgpu::less_t<uint>(), contexts[s]);});
					//t1.join();
				}
			}
			hipEventRecord(stop);

		} catch (mgpu::cuda_exception_t ex) {
			hipError_t errSync = hipGetLastError();
			hipError_t errAsync = hipDeviceSynchronize();
			if (errSync != hipSuccess)
				printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
			if (errAsync != hipSuccess)
				printf("Async kernel error: %s\n",
						hipGetErrorString(errAsync));
		}

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	cudaTest(hipMemcpy(h_vec, d_vec, mem_size_vec, hipMemcpyDeviceToHost));

	hipFree(d_vec);

	if (ELAPSED_TIME != 1) {
		//print(h_vec, num_of_segments, num_of_elements/num_of_segments);
		check_results(num_of_segments, num_of_elements / num_of_segments,
				h_vec);
	}

	free(h_seg);
	free(h_vec);

	return 0;
}

/***
 * SEGMENTED SORT FUNCIONANDO
 *
 *
 uint n = atoi(argv[1]);
 uint m = atoi(argv[2]);
 uint num_segments = n / m;
 mgpu::standard_context_t context;
 rand_key<uint> func(m);

 mgpu::mem_t<uint> segs = mgpu::fill_function(func, num_segments, context);
 //mgpu::mem_t<uint> segs = mgpu::fill_random(0, n - 1, num_segments, true, context);
 std::vector<uint> segs_host = mgpu::from_mem(segs);
 mgpu::mem_t<uint> data = mgpu::fill_random(0, pow(2, NUMBER_BITS_SIZE), n,
 false, context);
 mgpu::mem_t<uint> values(n, context);
 std::vector<uint> data_host = mgpu::from_mem(data);

 //	print(segs_host); print(data_host);

 mgpu::segmented_sort(data.data(), values.data(), n, segs.data(),
 num_segments, mgpu::less_t<uint>(), context);

 std::vector<uint> sorted = from_mem(data);
 std::vector<uint> indices_host = from_mem(values);

 std::cout << "\n";
 //print(segs_host);
 //	print(data_host); print(indices_host);
 *
 */
