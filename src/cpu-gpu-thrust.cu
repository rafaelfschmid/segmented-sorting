/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/system/cuda/detail/par.h>

//#include <future>
//#include <thread>

#include <algorithm>
#include <iostream>
#include <omp.h>
#include <vector>

#include <cudaProfiler.h>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 32
#endif

void print(uint* h_vec, int n) {
	std::cout << "\n";
	for (int i = 0; i < n; i++) {
		std::cout << h_vec[i] << " ";
	}
	std::cout << "\n";
}

//template<class T>
void kernelCall(thrust::system::cuda::detail::execute_on_stream exec, thrust::detail::normal_iterator<thrust::device_ptr<uint>> first, thrust::detail::normal_iterator<thrust::device_ptr<uint>> last){
//void kernelCall(thrust::cuda_cub::execute_on_stream exec, thrust::detail::normal_iterator<thrust::device_ptr<uint>> first, thrust::detail::normal_iterator<thrust::device_ptr<uint>> last){
	thrust::sort(exec,first,last);
}

int main(void) {
	int num_of_segments;
	int num_of_elements;
	int i;

	scanf("%d", &num_of_segments);
	uint mem_size_seg = sizeof(uint) * (num_of_segments + 1);
	uint *h_seg = (uint *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	uint mem_size_vec = sizeof(uint) * num_of_elements;
	uint *h_vec_aux = (uint *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++)
		scanf("%d", &h_vec_aux[i]);

	hipStream_t streams[NUM_STREAMS];
	for(int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&streams[i]);
		//hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
	}

	uint* d_vec;
	hipMallocManaged((void **)&d_vec, sizeof(uint)*num_of_elements);

	int nstreams = NUM_STREAMS;
	if(NUM_STREAMS > num_of_segments)
		nstreams = num_of_segments;

	omp_lock_t semaphore_lock;
	omp_init_lock(&semaphore_lock);
	
	for (uint i = 0; i < EXECUTIONS; i++) {
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		for (i = 0; i < num_of_elements; i++)
			d_vec[i] = h_vec_aux[i];

		hipProfilerStart();
		hipEventRecord(start);

		omp_set_num_threads(nstreams);
		int s = 0;
		#pragma omp parallel
		{
			uint id = omp_get_thread_num(); //cpu_thread_id

			if(id < 0){
				while(true) {
					omp_set_lock(&semaphore_lock);
					uint k = s;
					s++;
					omp_unset_lock(&semaphore_lock);

					if(k >= num_of_segments) {
						break;
					}
					//printf("i=%d   ---   k=%d\n", s, k);

					//thrust::sort(thrust::cuda::par.on(streams[id]), d_vec.begin() + h_seg[k], d_vec.begin() + h_seg[k + 1]);
					thrust::sort(thrust::cuda::par.on(streams[id]), d_vec + h_seg[k], d_vec + h_seg[k + 1]);
				}
			}
			else {
				while(true) {
					omp_set_lock(&semaphore_lock);
					uint k = s;
					s++;
					omp_unset_lock(&semaphore_lock);

					if(k >= num_of_segments) {
						break;
					}
					//printf("i=%d   ---   k=%d\n", s, k);

					std::stable_sort(&d_vec[h_seg[k]], &d_vec[h_seg[i+1]]);
				}
			}
		}

		hipEventRecord(stop);
		hipProfilerStop();

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	if (ELAPSED_TIME != 1) {
		print(d_vec, num_of_elements);
	}

	hipFree(streams);

	return 0;
}


/*thrust::sort(thrust::cuda::par.on(streams[0]), d_vec.begin() + h_seg[i],
		d_vec.begin() + h_seg[i + 1]);
thrust::sort(thrust::cuda::par.on(streams[1]), d_vec.begin() + h_seg[i+1],
		d_vec.begin() + h_seg[i + 2]);

thrust::sort(thrust::cuda::par.on(streams[2]), d_vec.begin() + h_seg[i+2],
		d_vec.begin() + h_seg[i + 3]);*/
//thrust::sort(thrust::cuda::par.on(streams[3]), d_vec.begin() + h_seg[i+3],d_vec.begin() + h_seg[i + 4]);
//std::async(std::launch::async, &kernelCall,streams[0], d_vec, h_seg, i);
//std::thread t1(&kernelCall<thrust::device_vector<uint>, thrust::host_vector<int>>,streams[0], d_vec, h_seg, i+0);
//std::thread t2(&kernelCall<thrust::device_vector<uint>, thrust::host_vector<int>>,streams[1], d_vec, h_seg, i+1);
//std::thread t3(&kernelCall<thrust::device_vector<uint>, thrust::host_vector<int>>,streams[2], d_vec, h_seg, i+2);
//std::thread t4(&kernelCall<thrust::device_vector<uint>, thrust::host_vector<int>>,streams[3], d_vec, h_seg, i+3);
//t1.join();
//t2.join();
//t3.join();
//t4.join();

//std::async(kernelCall,streams[1], d_vec, h_seg, i+1);
//std::async(kernelCall,streams[2], d_vec, h_seg, i+2);
//std::async(kernelCall,streams[3], d_vec, h_seg, i+3);
