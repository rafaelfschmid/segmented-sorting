#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define MAX_DEPTH       16
#define INSERTION_SORT  32

#ifndef NUM_STREAMS
#define NUM_STREAMS 16
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(uint* host_data, uint n, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << host_data[i * m + j] << " ";
		}
		std::cout << "\n";
	}

}

////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort(unsigned int *data, int left, int right)
{
	//printf("left=%d right=%d \n", left, right);
    for (int i = left ; i <= right ; ++i)
    {
        unsigned min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            unsigned val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(unsigned int *data, int left, int right, int depth)
{
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }

    unsigned int *lptr = data+left;
    unsigned int *rptr = data+right;
    unsigned int  pivot = data[(left+right)/2];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        // Find the next left- and right-hand values to swap
        unsigned int lval = *lptr;
        unsigned int rval = *rptr;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Call the quicksort kernel from the host.
////////////////////////////////////////////////////////////////////////////////
void run_qsort(unsigned int *data, unsigned int nitems)
{
    // Prepare CDP for the max depth 'MAX_DEPTH'.
    checkCudaErrors(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

    int left = 0;
    int right = nitems-1;
//    std::cout << "Launching kernel on the GPU" << std::endl;
    cdp_simple_quicksort<<< 1, 1 >>>(data, left, right, 0);
    checkCudaErrors(hipDeviceSynchronize());
}

////////////////////////////////////////////////////////////////////////////////
// Verify the results.
////////////////////////////////////////////////////////////////////////////////
void check_results(int n, int m, unsigned int *results_h)
{
    for (int i = 0 ; i < n ; ++i) {
    	for (uint j = 1; j < m; j++) {
    		if (results_h[i*m +j -1] > results_h[i*m +j])
			{
				std::cout << "Invalid item[" << j-1 << "]: " << results_h[i*m +j -1] << " greater than " << results_h[i*m +j] << std::endl;
				exit(EXIT_FAILURE);
			}
    	}
    }

    std::cout << "OK" << std::endl;
}

////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	uint num_of_segments;
	uint num_of_elements;
	uint i;

	scanf("%d", &num_of_segments);
	uint mem_size_seg = sizeof(uint) * (num_of_segments+1);
	uint *h_seg = (uint *) malloc(mem_size_seg);
	for (i = 0; i <= num_of_segments; i++) {
		scanf("%d", &h_seg[i]);
	}

	scanf("%d", &num_of_elements);
	uint mem_size_vec = sizeof(uint) * num_of_elements;
	uint *h_vec = (uint *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++){
		scanf("%d", &h_vec[i]);
	}

	hipStream_t streams[NUM_STREAMS];
	for(int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&streams[i]);
	}

	int nstreams = NUM_STREAMS;
	if(NUM_STREAMS > num_of_segments)
		nstreams = num_of_segments;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_vec;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	//print(h_vec, num_of_segments,num_of_elements/num_of_segments);

	for (int i = 0; i < EXECUTIONS; i++) {

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));

		hipEventRecord(start);
		for(int j = 0; j < num_of_segments; j+=nstreams) {
			//run_qsort(d_vec+h_seg[j], num_of_elements/num_of_segments);
			for(int s = 0; s < nstreams; s++) {
				cdp_simple_quicksort<<< 1, 1, 0, streams[s] >>>(d_vec, h_seg[j+s], h_seg[j+1+s]-1, 0);
			}

		}
		hipEventRecord(stop);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	hipMemcpy(h_vec, d_vec, mem_size_vec, hipMemcpyDeviceToHost);

	hipFree(d_vec);

	if (ELAPSED_TIME != 1) {
		//print(h_vec, num_of_segments, num_of_elements/num_of_segments);
		check_results(num_of_segments, num_of_elements/num_of_segments, h_vec);
	}

	free(h_vec);

	return 0;
}



