/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

//#include <future>
#include <thread>
#include <chrono>
#include <iostream>
#include <omp.h>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 4
#endif

void print(thrust::host_vector<int> h_vec) {
	std::cout << "\n";
	for (int i = 0; i < h_vec.size(); i++) {
		std::cout << h_vec[i] << " ";
	}
	std::cout << "\n";
}

template<typename D, typename H>
void kernelCall(hipStream_t stream, D d_vec, H h_seg, int i){

	hipSetDevice(0);
	printf("teste segment %d\n", i);
	thrust::sort(thrust::cuda::par.on(stream),d_vec.begin() + h_seg[i], d_vec.begin() + h_seg[i + 1]);


}

int main(void) {
	int num_of_segments;
	int num_of_elements;
	int i;

	scanf("%d", &num_of_segments);
	thrust::host_vector<int> h_seg(num_of_segments + 1);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	thrust::host_vector<int> h_vec(num_of_elements);
	for (i = 0; i < num_of_elements; i++)
		scanf("%d", &h_vec[i]);

	thrust::device_vector<uint> d_vec(num_of_elements);

	hipStream_t streams[NUM_STREAMS];
	for(int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&streams[i]);
		//hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
	}

	int nstreams = NUM_STREAMS;
	if(NUM_STREAMS > num_of_segments)
		nstreams = num_of_segments;

	omp_set_num_threads(nstreams);

	for (uint i = 0; i < EXECUTIONS; i++) {
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

		hipEventRecord(start);
		#pragma omp parallel
		{
			uint id = omp_get_thread_num(); //cpu_thread_id
			for (int i = 0; i < num_of_segments; i+=nstreams) {
				uint k = i + id;
				thrust::sort(thrust::cuda::par.on(streams[id]), d_vec.begin() + h_seg[k], d_vec.begin() + h_seg[k + 1]);
			}
		}
		hipEventRecord(stop);

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	if (ELAPSED_TIME != 1) {
		print(h_vec);
	}

	hipFree(streams);

	return 0;
}


/*thrust::sort(thrust::cuda::par.on(streams[0]), d_vec.begin() + h_seg[i],
		d_vec.begin() + h_seg[i + 1]);
thrust::sort(thrust::cuda::par.on(streams[1]), d_vec.begin() + h_seg[i+1],
		d_vec.begin() + h_seg[i + 2]);

thrust::sort(thrust::cuda::par.on(streams[2]), d_vec.begin() + h_seg[i+2],
		d_vec.begin() + h_seg[i + 3]);*/
//thrust::sort(thrust::cuda::par.on(streams[3]), d_vec.begin() + h_seg[i+3],d_vec.begin() + h_seg[i + 4]);
//std::async(std::launch::async, &kernelCall,streams[0], d_vec, h_seg, i);
//std::thread t1(&kernelCall<thrust::device_vector<uint>, thrust::host_vector<int>>,streams[0], d_vec, h_seg, i+0);
//std::thread t2(&kernelCall<thrust::device_vector<uint>, thrust::host_vector<int>>,streams[1], d_vec, h_seg, i+1);
//std::thread t3(&kernelCall<thrust::device_vector<uint>, thrust::host_vector<int>>,streams[2], d_vec, h_seg, i+2);
//std::thread t4(&kernelCall<thrust::device_vector<uint>, thrust::host_vector<int>>,streams[3], d_vec, h_seg, i+3);
//t1.join();
//t2.join();
//t3.join();
//t4.join();

//std::async(kernelCall,streams[1], d_vec, h_seg, i+1);
//std::async(kernelCall,streams[2], d_vec, h_seg, i+2);
//std::async(kernelCall,streams[3], d_vec, h_seg, i+3);
